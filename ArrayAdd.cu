#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "ArrayAdd.cuh"


__global__ void addKernel1D(float *dst, float *src1, float *src2, int n_data)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i < n_data) { dst[i] = src1[i] + src2[i]; }
}

bool arrayAddCuda(float* const dst, float *const src1, float *const src2, int const n_data, int const cuda_dev_id)
{
	hipError_t cudaStatus = hipSuccess;

	// set cuda device;
	cudaStatus = hipSetDevice(cuda_dev_id);
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Unable to set CUDA device " << cuda_dev_id << ", " << hipGetErrorString(cudaStatus) <<"." << std::endl;
		hipDeviceReset();
		return false;
	}

	// Allocate GPU buffers for three vectors
	float *_d_src1 = nullptr;
	float *_d_src2 = nullptr;
	float *_d_dst = nullptr;
	cudaStatus = hipMalloc((void**)&_d_dst, n_data * sizeof(float));
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Failed to allocate CUDA memory, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}
	cudaStatus = hipMalloc((void**)&_d_src1, n_data * sizeof(float));
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Failed to allocate CUDA memory, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}
	cudaStatus = hipMalloc((void**)&_d_src2, n_data * sizeof(float));
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Failed to allocate CUDA memory, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(_d_src1, src1, n_data * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Failed to copy memory from host to device, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}
	cudaStatus = hipMemcpy(_d_src2, src2, n_data * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Failed to copy memory from host to device, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}

	// determine thread size and block size
	int const max_thread_size = 1024;
	int block_dim = 0;
	int grid_dim = 0;
	if (n_data <= max_thread_size)
	{
		block_dim = n_data;
		grid_dim = 1;
	}
	else
	{
		block_dim = max_thread_size;
		grid_dim = n_data / max_thread_size + (int)((n_data%max_thread_size) > 0);
	}
	// Launch a kernel on the GPU with one thread for each element.
	addKernel1D << <grid_dim, block_dim >> >(_d_dst, _d_src1, _d_src2, n_data);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Failed to perform addition, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Failed to synchronized, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dst, _d_dst, n_data * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Failed to copy memory from host to device, " << hipGetErrorString(cudaStatus) << "." << std::endl;
		hipDeviceReset();
		return false;
	}
	hipFree(_d_src1);
	hipFree(_d_src2);
	hipFree(_d_dst);
	hipDeviceReset();
	return true;
}


void arrayAdd(float* const dst, float* const src1, float* const src2, int const n_data)
{// cpu
	for (int i = 0; i < n_data; i++) { dst[i] = src1[i] + src2[i]; }
}
